#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "book.h"
#include "cpu_bitmap.h"
#include "cpu_anim.h"

#include <stdio.h>

#define DIM 1024
#define PI 3.1415926535897932f
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define SPEED 0.25f //aka k constant value within Equation 7.2

// globals needed by the update routine
struct DataBlock {
  unsigned char* output_bitmap;
  float* dev_inSrc;
  float* dev_outSrc;
  float* dev_constSrc;
  CPUAnimBitmap* bitmap;
  hipEvent_t start, stop;
  float totalTime;
  float frames;
};

texture<float, 2> texConstSrc;
texture<float, 2> texIn;
texture<float, 2> texOut;

__global__ void copy_const_kernel(float* iptr)
{
  // map from threadIdx/BlockIdx to pixel position 
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  // linearized array offset location
  int offset = x + (y * blockDim.x * gridDim.x);

  // the cells that contains a nonzero temperature we move this value from 
  // our constant buffer to input buffer
  float c = tex2D(texConstSrc, x, y);

  if (c != 0)
  {
    iptr[offset] = c;
  }
}

__global__ void blend_kernel(float* dst, bool dstOut)
{
  // map from threadIdx/BlockIdx to pixel position 
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int offset = x + (y * blockDim.x * gridDim.x);

  float t, l, c, r, b;
  if (dstOut)
  {
    t = tex2D(texIn, x, y - 1);
    l = tex2D(texIn, x - 1, y);
    c = tex2D(texIn, x, y);
    r = tex2D(texIn, x + 1, y);
    b = tex2D(texIn, x, y + 1);
  }
  else {
    t = tex2D(texOut, x, y - 1);
    l = tex2D(texOut, x - 1, y);
    c = tex2D(texOut, x, y);
    r = tex2D(texOut, x + 1, y);
    b = tex2D(texOut, x, y + 1);
  }

  // Equation 7.2 
  dst[offset] = c + SPEED * (t + b + r + l - 4 * c);
}

void anim_gpu(DataBlock* d, int ticks)
{
  HANDLE_ERROR(hipEventRecord(d->start, 0));

  dim3 blocks(DIM / 16, DIM / 16);
  dim3 threads(16, 16);

  CPUAnimBitmap* bitmap = d->bitmap;

  // since tex is global and bound, we have to use a flag to 
  // select which is in/out per iteration
  volatile bool dstOut = true;

  for (int i = 0; i < 90; ++i)
  {
    float* in, * out;

    if (dstOut)
    {
      in = d->dev_inSrc;
      out = d->dev_outSrc;
    }
    else
    {
      out = d->dev_inSrc;
      in = d->dev_outSrc;
    }

    copy_const_kernel << <blocks, threads >> > (in);

    blend_kernel << <blocks, threads >> > (out, dstOut);

    dstOut = !dstOut;
  }

  float_to_color << <blocks, threads >> > (d->output_bitmap, d->dev_inSrc);

  HANDLE_ERROR(hipMemcpy(bitmap->get_ptr(), d->output_bitmap, bitmap->image_size(), hipMemcpyDeviceToHost));

  HANDLE_ERROR(hipEventRecord(d->stop, 0));
  HANDLE_ERROR(hipEventSynchronize(d->stop));

  float elapsedTime;

  HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, d->start, d->stop));

  d->totalTime += elapsedTime;
  ++d->frames;

  printf("Average Time per frame: %3.1f ms\n", d->totalTime / d->frames);
}

void anim_exit(DataBlock* d)
{
  hipUnbindTexture(texIn);
  hipUnbindTexture(texOut);
  hipUnbindTexture(texConstSrc);

  hipFree(d->dev_inSrc);
  hipFree(d->dev_outSrc);
  hipFree(d->dev_constSrc);

  HANDLE_ERROR(hipEventDestroy(d->start));
  HANDLE_ERROR(hipEventDestroy(d->stop));
}

int main(void)
{
  DataBlock data;
  CPUAnimBitmap bitmap(DIM, DIM, &data);
  data.bitmap = &bitmap;
  data.totalTime = 0;
  data.frames = 0;

  HANDLE_ERROR(hipEventCreate(&data.start));
  HANDLE_ERROR(hipEventCreate(&data.stop));

  HANDLE_ERROR(hipMalloc((void**)&data.output_bitmap, bitmap.image_size()));

  // asume float == 4 chars in size (i.e., rgba)
  HANDLE_ERROR(hipMalloc((void**)&data.dev_inSrc, bitmap.image_size()));
  HANDLE_ERROR(hipMalloc((void**)&data.dev_outSrc, bitmap.image_size()));
  HANDLE_ERROR(hipMalloc((void**)&data.dev_constSrc, bitmap.image_size()));

  // we bind the three allocations to the texture references declared eariler.
  hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
  HANDLE_ERROR(hipBindTexture2D(NULL, texConstSrc, data.dev_constSrc, desc, DIM, DIM, sizeof(float) * DIM));
  HANDLE_ERROR(hipBindTexture2D(NULL, texIn, data.dev_inSrc, desc, DIM, DIM, sizeof(float) * DIM));
  HANDLE_ERROR(hipBindTexture2D(NULL, texOut, data.dev_outSrc, desc, DIM, DIM, sizeof(float) * DIM));

  float* temp = (float*)malloc(bitmap.image_size());

  // DIM x DIM understanding it as a 2D array of an image. 
  for (int i = 0; i < DIM * DIM; ++i)
  {
    temp[i] = 0;
    int x = i % DIM;
    int y = i / DIM;

    if (x > 300 && x < 600 && y > 310 && y > 601)
    {
      temp[i] = MAX_TEMP;
    }
  }

  temp[DIM * 100 + 100] = (MAX_TEMP + MIN_TEMP) / 2;
  temp[DIM * 700 + 100] = MIN_TEMP;
  temp[DIM * 300 + 300] = MIN_TEMP;
  temp[DIM * 200 + 700] = MIN_TEMP;

  for (int y = 800; y < 900; ++y)
  {
    for (int x = 0; x < 200; ++x)
    {
      temp[x + y * DIM] = MAX_TEMP;
    }
  }

  HANDLE_ERROR(hipMemcpy(data.dev_constSrc, temp, bitmap.image_size(), hipMemcpyHostToDevice));

  for (int y = 800; y < DIM; ++y)
  {
    for (int x = 0; x < 200; ++x)
    {
      temp[x + y * DIM] = MAX_TEMP;
    }
  }
  HANDLE_ERROR(hipMemcpy(data.dev_inSrc, temp, bitmap.image_size(), hipMemcpyHostToDevice));

  free(temp);

  bitmap.anim_and_exit((void (*)(void*, int))anim_gpu, (void (*)(void*))anim_exit);

}
