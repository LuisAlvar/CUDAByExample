#include "hip/hip_runtime.h"
#include "./common/book.h"

#define SIZE (100*1024*1024)

__global__ void histo_kernel(unsigned char* buffer, long size, unsigned int * histo);

int main( void ) {

  // Random input stream data
  unsigned char *buffer = (unsigned char*) big_random_block( SIZE );

  // Initialize events for timing our task
  hipEvent_t start, stop;
  
  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, 0));

  // allocate memory on the GPU for the file's data
  unsigned char *dev_buffer;
  unsigned int *dev_histo;

  HANDLE_ERROR(hipMalloc((void**)&dev_buffer, SIZE)); // allocation
  HANDLE_ERROR(hipMemcpy(dev_buffer, buffer, SIZE, hipMemcpyHostToDevice)); // data transfer: CPU -> GPU

  HANDLE_ERROR(hipMalloc((void**)&dev_histo, 256*sizeof(int))); // allocation
  HANDLE_ERROR(hipMemset(dev_histo, 0, 256*sizeof(int))); // initialization of GPU buffer

  // TODO: Kernel call function 
  hipDeviceProp_t prop;
  HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
  int blocks = prop.multiProcessorCount;
  printf("Num of MultiProcessors: %d\n", blocks);
  histo_kernel<<<blocks*2, 256>>>(dev_buffer, SIZE, dev_histo);

  unsigned int histo[256];
  HANDLE_ERROR(hipMemcpy(histo, dev_histo, 256* sizeof(int), hipMemcpyDeviceToHost)); // data transfer: GPU -> CPU

  // get stop time, and display the timing result
  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  float elapsedTime;
  HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
  printf("Time to generate: %3.1f ms\n");

  // Unit Test: Verification
  long histoCount = 0;
  for (int i = 0; i < 256; i++)
  {
    histoCount += histo[i];
  }
  printf("Histogram Sum: %ld\n", histoCount);
  
  // verify that we have the same counts via CPU
  for (int i = 0; i < SIZE; i++)
  {
    histo[buffer[i]]--;
  }
  for (int i = 0; i < 256; i++)
  {
    if (histo[i] != 0)
    {
      printf("Failure at %d\n", i);
    }
  }
  
  // Clean up
  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));
  hipFree(dev_histo);
  hipFree(dev_buffer);
  free(buffer);
  
  return 0;
}

__global__ void histo_kernel(unsigned char* buffer, long size, unsigned int * histo) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  while (i < size)  
  {
    atomicAdd(&(histo[buffer[i]]), 1);
    i += stride;
  }
  
}
