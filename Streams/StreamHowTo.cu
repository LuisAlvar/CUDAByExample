#include "./common/book.h"

int main( void ) {

  hipDeviceProp_t prop;
  int whichDevice;
  
  HANDLE_ERROR(hipGetDevice(&whichDevice));
  HANDLE_ERROR(hipGetDeviceProperties(&prop, whichDevice));

  if (!prop.deviceOverlap)
  {
    printf("Device will not handle overlaps, so no speed up from streams\n");
    return 0;
  }
  
  hipEvent_t start, stop;
  float elapsedTime;

  // stat the timers
  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, 0));

  // initialize the stream 
  hipStream_t stream;
  HANDLE_ERROR( hipStreamCreate(&stream) );

  //
  
}