#include "./common/book.h"

int main( void ) {

  hipDeviceProp_t prop;
  int whichDevice;
  
  HANDLE_ERROR(hipGetDevice(&whichDevice));
  HANDLE_ERROR(hipGetDeviceProperties(&prop, whichDevice));

  if (!prop.deviceOverlap)
  {
    printf("Device will not handle overlaps, so no speed up from streams\n");
    return 0;
  }
  
  hipEvent_t start, stop;
  float elapsedTime;

  // stat the timers
  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start, 0));

  // initialize the stream 
  hipStream_t stream;
  HANDLE_ERROR( hipStreamCreate(&stream) );

  int *host_a, *host_b,*host_c;
  int *dev_a, *dev_b, *dev_c;

  // allocate the memory on the GPU
  HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

  // allocate page-locked memory, used to stream
  HANDLE_ERROR(hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
  HANDLE_ERROR(hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
  HANDLE_ERROR(hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));

  for (int i = 0; i < FULL_DATA_SIZE; i++)
  {
    host_a[i] = rand();
    host_b[i] = rand();
  }
  
  // now loop over full data, in bite-sized chunks
  for (int i = 0; i < FULL_DATA_SIZE; i += N)
  {
    // copy the locked memory to the device, async
    HANDLE_ERROR(hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream));

    HANDLE_ERROR(hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream));

    kernel<<<N/256, 256,0>>>(dev_a, dev_b, dev_c);

    // copy the data from device to locked memory 
    HANDLE_ERROR(hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream));
  }
  
}