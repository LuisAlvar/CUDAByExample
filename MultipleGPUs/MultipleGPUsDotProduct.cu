#include "hip/hip_runtime.h"
#include <stdio.h>
#include "./common/book.h"

// nvcc -Llib ./MultipleGPUsDotProduct.cu -o ./bin/MultipleGPUsDotProduct.exe

#define imin(a,b) (a<b?a:b)
const int N = 33 * 1024 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N+threadsPerBlock-1)/threadsPerBlock);

void* routine(void *pvoidData);

struct DataStruct {
  int deviceId;
  int size;
  float *a;
  float *b;
  float returnValue;
};

__global__ void dot(int size, float *a, float *b, float *c) {
  __shared__ float cache[threadsPerBlock];

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int cacheIndex = threadIdx.x;
  float temp = 0;
  while (tid < size)
  {
    temp += a[tid] * b[tid];
    tid += blockDim.x * gridDim.x;

  }
  
  // set the cache valeus
  cache[cacheIndex] = temp;

  // synchronize threads in this block
  __syncthreads();

  // for reductions, threadsPerBlock must be power of 2
  // because of the following code
  int i = blockDim.x / 2;
  while (i != 0)
  {
    if (cacheIndex < i)
    {
      cache[cacheIndex] += cache[cacheIndex + i];
    }
    __syncthreads();
    i /= 2;
  }

  if (cacheIndex == 0)
  {
    c[blockIdx.x] = cache[0];
  }
}

int main ( void ) {

  int deviceCount;
  HANDLE_ERROR(hipGetDeviceCount(&deviceCount));
  if (deviceCount < 2)  
  {
    printf("We need at least two compute 1.0 or greater devices, but only found %d\n", deviceCount);
    return 0;
  }

  float *a = (float*) malloc(sizeof(float) * N);
  HANDLE_NULL(a);
  float *b = (float*) malloc(sizeof(float) * N);
  HANDLE_NULL(b);

  // fill in the host memory with data
  for( int i = 0; i < N; ++i)
  {
    a[i] = i;
    b[i] = i * 2;
  }
  
  DataStruct data[2];
  data[0].deviceId = 0;
  data[0].size = N/2;
  data[0].a = a;
  data[0].b = b;

  data[1].deviceId = 1;
  data[1].size = N/2;
  data[1].a = a + N/2;
  data[1].b = b + N/2;

  CUTThread thread = start_thread(routine, &(data[0]));
  routine(&(data[1]));

  // before we preoceed, we have the main applicaiton thread wait for the 
  // other thread to finsih by calling end_thread();
  end_thread( thread );

  free(a);
  free(b);

  printf("Value calculated: %f\n", data[0].returnValue + data[1].returnValue);

  return 0;
}

void* routine(void *pvoidData) {
  DataStruct *data = (DataStruct*)pvoidData;
  HANDLE_ERROR(hipSetDevice(data->deviceId));

  int size = data->size;
  float *a, *b, c, *partial_c;
  float *dev_a, *dev_b, *dev_partial_c;
  
  // allocate memory on the PCPU side
  a = data->a;
  b = data->b;
  partial_c = (float*)malloc(blocksPerGrid*sizeof(float));

  // allcoate memory on the GPU
  HANDLE_ERROR(hipMalloc((void**)&dev_a, size * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, size * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_partial_c, blocksPerGrid * sizeof(float)));

  // copy the arrays 'a' and 'b' to the GPU
  HANDLE_ERROR(hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice));

  dot<<<blocksPerGrid, threadsPerBlock>>>(size, dev_a, dev_b, dev_partial_c);

  // Copy the array 'c' back from the GPU to the CPU
  HANDLE_ERROR(hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost));

  // finish up on the cpu side
  c = 0;
  for (int i = 0; i < blocksPerGrid; i++)
  {
    c += partial_c[i];
  }

  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_partial_c));
  
  // free memory on the CPU side
  free(partial_c);

  data->returnValue = c;
  return 0;
}