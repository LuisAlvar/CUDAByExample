#include "hip/hip_runtime.h"
#include <stdio.h>
#include "./common/book.h"

// nvcc -Llib ./ZeroCopyDotProduct.cu -o ./bin/ZeroCopyDotProduct.exe
// First run 
// Value calculated: 27621697910970467221504.000000
// Time using hipMalloc: 25.6 ms
// Value calculated: 27621697910970467221504.000000
// Time using hipMalloc: 22.6 ms

#define imin(a,b) (a<b?a:b)
const int N = 33 * 1024 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N+threadsPerBlock-1)/threadsPerBlock);

float malloc_test(int size);
float cuda_host_alloc_test( int size );

__global__ void dot(int size, float *a, float *b, float *c) {
  __shared__ float cache[threadsPerBlock];

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int cacheIndex = threadIdx.x;
  float temp = 0;
  while (tid < size)
  {
    temp += a[tid] * b[tid];
    tid += blockDim.x * gridDim.x;

  }
  
  // set the cache valeus
  cache[cacheIndex] = temp;

  // synchronize threads in this block
  __syncthreads();

  // for reductions, threadsPerBlock must be power of 2
  // because of the following code
  int i = blockDim.x / 2;
  while (i != 0)
  {
    if (cacheIndex < i)
    {
      cache[cacheIndex] += cache[cacheIndex + i];
    }
    __syncthreads();
    i /= 2;
  }

  if (cacheIndex == 0)
  {
    c[blockIdx.x] = cache[0];
  }
}

int main( void ) {

  hipDeviceProp_t prop;
  int whichDevice;

  HANDLE_ERROR(hipGetDevice(&whichDevice));
  HANDLE_ERROR(hipGetDeviceProperties(&prop, whichDevice));

  if (prop.canMapHostMemory != 1)
  {
    printf("Device cannot map memory. \n");
    return 0;
  }

  // Runtime setting: indicates that we want the device to be allowed to map host 
  // memory. 
  HANDLE_ERROR(hipSetDeviceFlags(hipDeviceMapHost));
  
  // Perform two tests, display the elapsed time, and exit the application
  float elapsedTime = malloc_test(N);
  printf("Time using hipMalloc: %3.1f ms\n", elapsedTime);

  elapsedTime = cuda_host_alloc_test(N);
  printf("Time using hipHostAlloc: %3.1f ms\n", elapsedTime);

}



float malloc_test(int size) {

  hipEvent_t start, stop;

  float *a, *b, c, *partial_c;
  float *dev_a, *dev_b, *dev_partial_c;
  float elapsedTime;

  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));

  // allocate memory on the CPU side
  a = (float*)malloc(size * sizeof(float));
  b = (float*)malloc(size * sizeof(float));
  partial_c = (float*)malloc(blocksPerGrid * sizeof(float));

  // allocate the memory on the GPU
  HANDLE_ERROR(hipMalloc((void**)&dev_a, size * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, size * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_partial_c, blocksPerGrid * sizeof(float)));

  // fill in the host memory with data
  for(int i = 0; i < size; ++i) {
    a[i] = i;
    b[i] = i * 2;
  }

  // Start the time event 
  HANDLE_ERROR(hipEventRecord(start, 0));

  // copy the arrays 'a' and 'b' to the GPU
  HANDLE_ERROR(hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice));

  dot<<<blocksPerGrid, threadsPerBlock>>>(size, dev_a, dev_b, dev_partial_c);

  // copy the array 'c' back from the GPU to the CPU
  HANDLE_ERROR(hipMemcpy(partial_c, dev_partial_c, blocksPerGrid*sizeof(float), hipMemcpyDeviceToHost));

  // stop our event timer
  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

  // finish up on the CPU side
  // we sum our partial results and free our input and output buffers.
  c = 0;
  for (int i = 0; i < blocksPerGrid; i++)
  {
    c += partial_c[i];
  }
  
  // free memory on the GPU side
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_partial_c));

  // free memory on the CPU side
  free(a);
  free(b);
  free(partial_c);

  // free events
  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));

  printf("Value calculated: %f\n", c);

  return elapsedTime;
}


float cuda_host_alloc_test( int size ) {

  hipEvent_t start, stop;

  float *a, *b, c, *partial_c;
  float *dev_a, *dev_b, *dev_partial_c;
  float elapsedTime;

  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));

  // allocate the memory on the CPU 
  HANDLE_ERROR(hipHostAlloc((void**)&a, size * sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped));
  HANDLE_ERROR(hipHostAlloc((void**)&b, size * sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped));
  HANDLE_ERROR(hipHostAlloc((void**)&partial_c, blocksPerGrid * sizeof(float), hipHostMallocMapped));

  for (int i = 0; i < size; i++)
  {
    a[i] = i;
    b[i] = i * 2;
  }
  
  // CPU pointers to GPU pointers memory mapping 
  HANDLE_ERROR(hipHostGetDevicePointer(&dev_a, a, 0));
  HANDLE_ERROR(hipHostGetDevicePointer(&dev_b, b, 0));
  HANDLE_ERROR(hipHostGetDevicePointer(&dev_partial_c, partial_c, 0));

  // Start our timer and launch our kernel 
  HANDLE_ERROR(hipEventRecord(start, 0));
  dot<<<blocksPerGrid, threadsPerBlock>>>(size, dev_a, dev_b, dev_partial_c);
  HANDLE_ERROR(hipDeviceSynchronize());

  // After synchronizing, stop our timer and finsh the computation on the CPU as we did before. 
  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

  // finish up on the CPU side
  c = 0;
  for (int i = 0; i < blocksPerGrid; i++)
  {
    c += partial_c[i];
  }
  
  // clean up 
  HANDLE_ERROR(hipHostFree(a));
  HANDLE_ERROR(hipHostFree(b));
  HANDLE_ERROR(hipHostFree(partial_c));

  // free events
  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));

  printf("Value calculated: %f\n", c);

  return elapsedTime;
}