#include <stdio.h>
#include "./common/book.h"

float malloc_test(int size);
float cuda_host_alloc_test( int size );


int main( void ) {

  hipDeviceProp_t prop;
  int whichDevice;

  HANDLE_ERROR(hipGetDevice(&whichDevice));
  HANDLE_ERROR(hipGetDeviceProperties(&prop, whichDevice));

  if (prop.canMapHostMemory != 1)
  {
    printf("Device cannot map memory. \n");
    return 0;
  }
  
}



float malloc_test(int size) {

  hipEvent_t start, stop;

  float *a, *b, c, *partial_c;
  float *dev_a, *dev_b, *dev_partial_c;
  float elapsedTime;

  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));

  // allocate memory on the CPU side
  a = (float*)malloc(size * sizeof(float));
  b = (float*)malloc(size * sizeof(float));
  partial_c = (float*)malloc(blocksPerGrid * sizeof(float));

  // allocate the memory on the GPU
  HANDLE_ERROR(hipMalloc((void**)&dev_a, size * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, size * sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_partial_c, blocksPerGrid * sizeof(float)));

  // fill in the host memory with data
  for(int i = 0; i < size; ++i) {
    a[i] = i;
    b[i] = i * 2;
  }

  // Start the time event 
  HANDLE_ERROR(hipEventRecord(start, 0));

  // copy the arrays 'a' and 'b' to the GPU
  HANDLE_ERROR(hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice));

  dot<<<blocksPerGrid, threadsPerBlock>>>(size, dev_a, dev_b, dev_partial_c);

  // copy the array 'c' back from the GPU to the CPU
  HANDLE_ERROR(hipMemcpy(partial_c, dev_partial_c, blocksPerGrid*sizeof(float), hipMemcpyDeviceToHost));

  // stop our event timer
  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

  // finish up on the CPU side
  // we sum our partial results and free our input and output buffers.
  c = 0;
  for (int i = 0; i < blocksPerGrid; i++)
  {
    c += partial_c[i];
  }
  
  // free memory on the GPU side
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_partial_c));

  // free memory on the CPU side
  free(a);
  free(b);
  free(partial_c);

  // free events
  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));

  printf("Value calculated: %f\n", c);

  return elapsedTime;
}


float cuda_host_alloc_test( int size ) {

  hipEvent_t start, stop;

  float *a, *b, c, *partial_c;
  float *dev_a, *dev_b, *dev_partial_c;
  float elapsedTime;

  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));

  // allocate the memory on the CPU 
  HANDLE_ERROR(hipHostAlloc((void**)&a, size * sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped));
  HANDLE_ERROR(hipHostAlloc((void**)&b, size * sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped));
  HANDLE_ERROR(hipHostAlloc((void**)&partial_c, blocksPerGrid * sizeof(float), hipHostMallocMapped));

  for (int i = 0; i < size; i++)
  {
    a[i] = i;
    b[i] = i * 2;
  }
  
  // CPU pointers to GPU pointers memory mapping 
  HANDLE_ERROR(hipHostGetDevicePointer(&dev_a, a, 0));
  HANDLE_ERROR(hipHostGetDevicePointer(&dev_b, b, 0));
  HANDLE_ERROR(hipHostGetDevicePointer(&dev_partial_c, partial_c, 0));

  // Start our timer and launch our kernel 
  HANDLE_ERROR(hipEventRecord(start, 0));
  dot<<<blocksPerGrids, threadsPerBlock>>>(size, dev_a, dev_b, dev_partial_c);
  HANDLE_ERROR(hipDeviceSynchronize());

  // After synchronizing, stop our timer and finsh the computation on the CPU as we did before. 
  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

  // finish up on the CPU side
  c = 0;
  for (int i = 0; i < blocksPerGrid; i++)
  {
    c += partial_c[i];
  }
  
  // clean up 
  HANDLE_ERROR(hipHostFree(a));
  HANDLE_ERROR(hipHostFree(b));
  HANDLE_ERROR(hipHostFree(partial_c));

  // free events
  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));

  printf("Value calculated: %f\n", c);

  return elapsedTime;
}